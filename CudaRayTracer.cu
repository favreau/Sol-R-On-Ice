#include "hip/hip_runtime.h"
/* 
* OpenCL Raytracer
* Copyright (C) 2011-2012 Cyrille Favreau <cyrille_favreau@hotmail.com>
*
* This library is free software; you can redistribute it and/or
* modify it under the terms of the GNU Library General Public
* License as published by the Free Software Foundation; either
* version 2 of the License, or (at your option) any later version.
*
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
* Library General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

/*
* Author: Cyrille Favreau <cyrille_favreau@hotmail.com>
*
*/

// System
#include <iostream>

// Cuda
#include <hip/hip_runtime_api.h>
#if CUDART_VERSION>=5000
   #include <hip/hip_runtime_api.h>
   #include <hip/hip_vector_types.h>
#else
   #include <cutil_inline.h>
   #include <cutil_math.h>
#endif

// Project
#include "CudaDataTypes.h"

// Cuda Runtime paramters
#define BLOCK_SIZE 16

// Globals
#define gNbIterations 20
#define gNbMaxShadowCollisions 10
#define gSoftShadowPrecision   1.f

// Device arrays
Primitive*   d_primitives; 
BoundingBox* d_boundingBoxes; 
Lamp*        d_lamps;
Material*    d_materials;
char*        d_textures;
float*       d_randoms;
int*         d_levels;
float4*      d_postProcessingBuffer;
char*        d_bitmap; 

#ifdef USE_KINECT
char*        d_kinectVideo;
char*        d_kinectDepth;
#endif // USE_KINECT

// ________________________________________________________________________________
__device__ inline float vectorLength( float4 vector )
{
   return sqrt( vector.x*vector.x + vector.y*vector.y + vector.z*vector.z );
}

// ________________________________________________________________________________
__device__ inline void normalizeVector( float4& v )
{
   v /= vectorLength( v );
}

// ________________________________________________________________________________
__device__ inline void saturateVector( float4& v )
{
   v.x = (v.x>1.f) ? 1.f : v.x;
   v.y = (v.y>1.f) ? 1.f : v.y; 
   v.z = (v.z>1.f) ? 1.f : v.z;
   v.w = (v.w>1.f) ? 1.f : v.w;
}

// ________________________________________________________________________________
__device__ inline float dotProduct( float4 &v1, float4& v2 )
{
   return ( v1.x*v2.x + v1.y*v2.y + v1.z*v2.z);
}

/*
________________________________________________________________________________
incident  : le vecteur normal inverse a la direction d'incidence de la source 
lumineuse
normal    : la normale a l'interface orientee dans le materiau ou se propage le 
rayon incident
reflected : le vecteur normal reflechi
________________________________________________________________________________
*/
__device__ inline void vectorReflection( float4& r, float4& i, float4& n )
{
   r = i-2.f*dotProduct(i,n)*n;
}

/*
________________________________________________________________________________
incident: le vecteur norm? inverse ? la direction d?incidence de la source 
lumineuse
n1      : index of refraction of original medium
n2      : index of refraction of new medium
________________________________________________________________________________
*/
__device__ inline void vectorRefraction( 
   float4& refracted, 
   float4 incident, 
   float n1, 
   float4 normal, 
   float n2 )
{
   if( n1 != n2 ) 
   {
      float r = n1/n2;
      float cosI = dotProduct( incident, normal );
      float cosT2 = 1.0f - r*r*(1.0f - cosI*cosI);
      if( cosT2 > 0.01f ) 
      {
         refracted = r*incident + (r*cosI-sqrt( fabs(cosT2) ))*normal;
      }
   }
   else 
   {
      refracted = incident;
   }
}

/*
________________________________________________________________________________
__v : Vector to rotate
__c : Center of rotations
__a : Angles
________________________________________________________________________________
*/
__device__ inline void vectorRotation( float4& vector, float4 center, float4 angles )
{ 
   float4 result = vector; 
   /* X axis */ 
   result.y = vector.y*cos(angles.x) - vector.z*sin(angles.x); 
   result.z = vector.y*sin(angles.x) + vector.z*cos(angles.x); 
   vector = result; 
   result = vector; 
   /* Y axis */ 
   result.z = vector.z*cos(angles.y) - vector.x*sin(angles.y); 
   result.x = vector.z*sin(angles.y) + vector.x*cos(angles.y); 
   vector = result; 
}

/*
________________________________________________________________________________

Compute ray attributes
________________________________________________________________________________
*/
__device__ inline void computeRayAttributes(Ray& ray)
{
   ray.inv_direction.x = 1.f/ray.direction.x;
   ray.inv_direction.y = 1.f/ray.direction.y;
   ray.inv_direction.z = 1.f/ray.direction.z;
   ray.sign[0] = (ray.inv_direction.x < 0);
   ray.sign[1] = (ray.inv_direction.y < 0);
   ray.sign[2] = (ray.inv_direction.z < 0);
}

/*
________________________________________________________________________________

Convert float4 into OpenGL RGB color
________________________________________________________________________________
*/
__device__ void makeOpenGLColor( 
   float4 color,
   char*  bitmap,
   int    index)
{
   int mdc_index = index*gColorDepth; 
   bitmap[mdc_index  ] = (char)(color.x*255.f); // Red
   bitmap[mdc_index+1] = (char)(color.y*255.f); // Green
   bitmap[mdc_index+2] = (char)(color.z*255.f); // Blue
   bitmap[mdc_index+3] = (char)(color.w*255.f); // Alpha
}

/*
________________________________________________________________________________

Sphere texture Mapping
________________________________________________________________________________
*/
__device__ float4 sphereMapping( 
   Primitive& primitive,
   Material*  materials,
   char*      textures,
   float4     intersection)
{
   float4 result = materials[primitive.materialId].color;
   int x = gTextureOffset+(intersection.x-primitive.p0.x+primitive.size.y)*primitive.materialInfo.x;
   int y = gTextureOffset+(intersection.y-primitive.p0.y+primitive.size.x)*primitive.materialInfo.y;

   x = x % gTextureWidth;
   y = y % gTextureHeight;

   if( x>=0 && x<gTextureWidth&& y>=0 && y<gTextureHeight )
   {
      int index = (materials[primitive.materialId].textureId*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
      unsigned char r = textures[index  ];
      unsigned char g = textures[index+1];
      unsigned char b = textures[index+2];
      result.x = r/256.f;
      result.y = g/256.f;
      result.z = b/256.f;
   }
   return result; 
}

/*
________________________________________________________________________________

Cube texture mapping
________________________________________________________________________________
*/
__device__ float4 cubeMapping( 
   Primitive& primitive, 
   Material*  materials,
   char*      textures,
   float4     intersection)
{
   float4 result = materials[primitive.materialId].color;
   int x = ((primitive.type == ptCheckboard) || (primitive.type == ptXZPlane) || (primitive.type == ptXYPlane))  ? 
      gTextureOffset+(intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x :
   gTextureOffset+(intersection.z-primitive.p0.z+primitive.size.x)*primitive.materialInfo.x;

   int y = ((primitive.type == ptCheckboard) || (primitive.type == ptXZPlane)) ? 
      gTextureOffset+(intersection.z+primitive.p0.z+primitive.size.y)*primitive.materialInfo.y :
   gTextureOffset+(intersection.y-primitive.p0.y+primitive.size.y)*primitive.materialInfo.y;

   x = x%gTextureWidth;
   y = y%gTextureHeight;

   if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
   {
      int index = (materials[primitive.materialId].textureId*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
      unsigned char r = textures[index];
      unsigned char g = textures[index+1];
      unsigned char b = textures[index+2];
      result.x = r/256.f;
      result.y = g/256.f;
      result.z = b/256.f;
   }
   return result;
}

#if 0
/*
________________________________________________________________________________

Magic Carpet texture mapping
________________________________________________________________________________
*/
__device__ float4 magicCarpetMapping( 
   Primitive primitive, 
   Material* materials,
   char*     textures,
   float4    intersection,
   int*      levels,
   float     timer)
{
   float4 result = materials[primitive.materialId].color;
   int x = gTextureOffset+(intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x*5.f;
   int y = gTextureOffset+(intersection.z+timer-primitive.p0.z+primitive.size.y)*primitive.materialInfo.y*50.f;

   x = x%gTextureWidth;
   y = y%gTextureHeight;

   if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
   {
      // Level management
      int tid_x = (intersection.x-primitive.p0.x+primitive.size.x      )/(primitive.size.x/2.5f);
      int tid_y = (intersection.z-primitive.p0.z+primitive.size.y+timer)/(primitive.size.y/25.f);
      int tid = tid_x+tid_y*5;
      tid = tid%5000;
      int index = (levels[tid]*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
      unsigned char r = textures[index];
      unsigned char g = textures[index+1];
      unsigned char b = textures[index+2];
      result.x = r/256.f;
      result.y = g/256.f;
      result.z = b/256.f;
   }
   return result;
}

/*
________________________________________________________________________________

Magic Cylinder texture mapping
________________________________________________________________________________
*/
__device__ float4 magicCylinderMapping( 
   Primitive primitive, 
   Material* materials,
   char*     textures,
   float4    intersection,
   int*      levels,
   float     timer)
{
   float4 result = materials[primitive.materialId].color;

   int x = gTextureOffset+(intersection.x-      primitive.p0.x+primitive.size.x)*primitive.materialInfo.x*5.f;
   int y = gTextureOffset+(intersection.z+timer-primitive.p0.z+primitive.size.y)*primitive.materialInfo.y*50.f;

   x = x%gTextureWidth;
   y = y%gTextureHeight;

   if( x>=0 && x<gTextureWidth && y>=0 && y<gTextureHeight )
   {
      int tid_x = (intersection.x-primitive.p0.x+primitive.size.x      )/(primitive.size.x/2.5f);
      int tid_y = (intersection.z-primitive.p0.z+primitive.size.y+timer)/(primitive.size.y/25.f);
      int tid = tid_x+tid_y*5;
      tid = tid%5000;
      int index = (levels[tid]*gTextureWidth*gTextureHeight + y*gTextureWidth+x)*gTextureDepth;
      unsigned char r = textures[index  ];
      unsigned char g = textures[index+1];
      unsigned char b = textures[index+2];
      result.x = r/256.f;
      result.y = g/256.f;
      result.z = b/256.f;
   }
   return result;
}
#endif // 0

/*
________________________________________________________________________________

Box intersection
________________________________________________________________________________
*/
__device__ inline bool boxIntersection( 
   BoundingBox& box, 
   Ray          ray,
   float        t0,
   float        t1)
{
   computeRayAttributes( ray );

   float tmin, tmax, tymin, tymax, tzmin, tzmax;

   tmin = (box.parameters[ray.sign[0]].x - ray.origin.x) * ray.inv_direction.x;
   tmax = (box.parameters[1-ray.sign[0]].x - ray.origin.x) * ray.inv_direction.x;
   tymin = (box.parameters[ray.sign[1]].y - ray.origin.y) * ray.inv_direction.y;
   tymax = (box.parameters[1-ray.sign[1]].y - ray.origin.y) * ray.inv_direction.y;

   if ( (tmin > tymax) || (tymin > tmax) ) 
      return false;

   if (tymin > tmin) tmin = tymin;
   if (tymax < tmax) tmax = tymax;
   tzmin = (box.parameters[ray.sign[2]].z - ray.origin.z) * ray.inv_direction.z;
   tzmax = (box.parameters[1-ray.sign[2]].z - ray.origin.z) * ray.inv_direction.z;

   if ( (tmin > tzmax) || (tzmin > tmax) ) 
      return false;

   if (tzmin > tmin) tmin = tzmin;
   if (tzmax < tmax) tmax = tzmax;
   return ( (tmin < t1) && (tmax > t0) );
}

/*
________________________________________________________________________________

Lamp intersection
________________________________________________________________________________
*/
__device__ inline bool lampIntersection( 
   Lamp&   lamp, 
   Ray&    ray, 
   float4  O_C,
   float4& intersection)
{
   float si_A = 2.f*(ray.direction.x*ray.direction.x + ray.direction.y*ray.direction.y + ray.direction.z*ray.direction.z);
   if ( si_A == 0.f ) return false;

   bool  si_b1 = false; 
   float si_B = 2.f*(O_C.x*ray.direction.x + O_C.y*ray.direction.y + O_C.z*ray.direction.z);
   float si_C = O_C.x*O_C.x+O_C.y*O_C.y+O_C.z*O_C.z-lamp.center.w*lamp.center.w;
   float si_radius = si_B*si_B-2.f*si_A*si_C;
   float si_t1 = (-si_B-sqrt(si_radius))/si_A;

   if( si_t1>0.f ) 
   {
      intersection = ray.origin+si_t1*ray.direction;
      si_b1 = true;
   }
   return si_b1;
}

/*
________________________________________________________________________________

Sphere intersection
________________________________________________________________________________
*/
__device__ inline bool sphereIntersection(
   SceneInfo& sceneInfo,
   Primitive& sphere, 
   Material*  materials, 
   char*      textures, 
   Ray&       ray, 
   float      timer,
   float4&    intersection,
   float4&    normal,
   float&     shadowIntensity
   ) 
{
   // solve the equation sphere-ray to find the intersections
   bool result = false;

   float4 O_C = ray.origin - sphere.p0;
   normalizeVector(ray.direction);
   if(( dotProduct( O_C, ray.direction ) > 0.f ) && (vectorLength(O_C) > sphere.size.x)) return false;

   float a = 2.f*dotProduct(ray.direction,ray.direction);
   float b = 2.f*dotProduct(O_C,ray.direction);
   float c = dotProduct(O_C,O_C) - (sphere.size.x*sphere.size.x);
   float d = b*b-2.f*a*c;
   if( d>0.f && a != 0.f) 
   {
      float r = sqrt(d);
      float t1 = (-b-r)/a;
      float t2 = (-b+r)/a;
      float ta = (t1<t2) ? t1 : t2;
      float tb = (t2<t1) ? t1 : t2;
      float4 intersection1;
      float4 intersection2;
      bool i1(false);
      bool i2(false);

      if( ta > 0.1f ) 
      {
         // First intersection
         intersection1 = ray.origin+ta*ray.direction;
         // Transparency
         if (materials[sphere.materialId].textureId != NO_TEXTURE && materials[sphere.materialId].transparency != 0 ) 
         {
            float4 color = sphereMapping(sphere, materials, textures, intersection1 );
            i1 = ((color.x+color.y+color.z) >= sceneInfo.transparentColor ); 
         }
         else
         {
            i1 = true;
         }
      }

      if( tb > 0.1f ) 
      {
         // Second intersection
         intersection2 = ray.origin+tb*ray.direction;
         if (materials[sphere.materialId].textureId != NO_TEXTURE && materials[sphere.materialId].transparency != 0 ) 
         {
            float4 color = sphereMapping(sphere, materials, textures, intersection2 );
            i2 = ((color.x+color.y+color.z) >= sceneInfo.transparentColor ); 
         }
         else
         {
            i2 = true;
         }
      }

      result = i1 || i2;

      if( result ) 
      { 
         if( i1 && i2 )
         {
            float4 O_I1 = intersection1 - ray.origin;
            float4 O_I2 = intersection2 - ray.origin;
            intersection = ( vectorLength(O_I1)<vectorLength(O_I2)) ? intersection1 : intersection2;
         }
         else 
         {
            intersection = i1 ? intersection1 : intersection2;
         }

         // Compute normal vector
         normal = intersection-sphere.p0;
         normal.w = 0.f;
         shadowIntensity = sceneInfo.shadowIntensity*1.f-materials[sphere.materialId].transparency;

         if( materials[sphere.materialId].textured ) 
         {
            // Procedural texture
            float4 newCenter;
            newCenter.x = sphere.p0.x + 5.f*cos(intersection.x);
            newCenter.y = sphere.p0.y + 5.f*sin(intersection.y);
            newCenter.z = sphere.p0.z + 5.f*sin(cos(intersection.z));
            normal  = intersection - newCenter;
         }

         //float4 loi = intersection-origin;
         //if( dotProduct(loi,normal) <= 0.f ) normal = -normal;
         normalizeVector(normal);
      }
   }

#if 0
   // Soft Shadows
   if( result && computingShadows ) 
   {
      float4 O_R;
      O_R.x = ray.x-origin.x;
      O_R.y = ray.y-origin.y;
      O_R.z = ray.z-origin.z;

      normalizeVector(O_R);
      shadowIntensity = dotProduct(O_R, normal);
      shadowIntensity = (shadowIntensity>1.f) ? 1.f : shadowIntensity;
      shadowIntensity = (shadowIntensity<0.f) ? 0.f : shadowIntensity;
   } 
#endif // 0
   return result;
}

/*
________________________________________________________________________________

Cylinder intersection
________________________________________________________________________________
*/
__device__ bool cylinderIntersection( 
   SceneInfo& sceneInfo,
   Primitive& cylinder,
   Material* materials, 
   char*     textures,
   int*      levels,
   Ray       ray, 
   float     timer,
   float4&   intersection,
   float4&   normal,
   float&    shadowIntensity ) 
{
   // solve the equation sphere-ray to find the intersections
   bool result = false;

   /*
   // Top
   if(!result && ray.y<0.f && origin.y>(cylinder.p0.y+cylinder.size.y)) 
   {
      intersection.y = cylinder.p0.y+cylinder.size.y;
      float y = origin.y-cylinder.p0.y-cylinder.size.y;
      intersection.x = origin.x+y*ray.x/-ray.y;
      intersection.z = origin.z+y*ray.z/-ray.y;
      intersection.w = 1.f; // 1 for top, -1 for bottom

      float4 v=intersection-cylinder.p0;
      v.y = 0.f;
      result = (vectorLength(v)<cylinder.size.x);

      normal.x =  0.f;
      normal.y =  1.f;
      normal.z =  0.f;
   }

   // Bottom
   if( !result && ray.y>0.f && origin.y<(cylinder.p0.y - cylinder.size.y) ) 
   {
      intersection.y = cylinder.p0.y - cylinder.size.y;
      float y = origin.y - cylinder.p0.y + cylinder.size.y;
      intersection.x = origin.x+y*ray.x/-ray.y;
      intersection.z = origin.z+y*ray.z/-ray.y;
      intersection.w = -1.f; // 1 for top, -1 for bottom

      float4 v=intersection-cylinder.p0;
      v.y = 0.f;
      result = (vectorLength(v)<cylinder.size.x);

      normal.x =  0.f;
      normal.y = -1.f;
      normal.z =  0.f;
   }
   */

   if( !result ) 
   {
      float4 O_C = ray.origin - cylinder.p0;
      O_C.y = 0.f;
      if(( dotProduct( O_C, ray.direction ) > 0.f ) && (vectorLength(O_C) > cylinder.p0.w)) return false;

      float a = 2.f * ( ray.direction.x*ray.direction.x + ray.direction.z*ray.direction.z );
      float b = 2.f*((ray.origin.x-cylinder.p0.x)*ray.direction.x + (ray.origin.z-cylinder.p0.z)*ray.direction.z);
      float c = O_C.x*O_C.x + O_C.z*O_C.z - cylinder.size.y*cylinder.size.y;

      float d = b*b-2.f*a*c;

      // Cylinder
      if ( /*d >= 0.f &&*/ a != 0.f) 
      {
         float r = sqrt(d);
         float t1 = (-b-r)/a;
         float t2 = (-b+r)/a;
         float ta = (t1<t2) ? t1 : t2;
         float tb = (t2<t1) ? t1 : t2;

         float4 intersection1;
         float4 intersection2;
         bool i1(false);
         bool i2(false);

         if( ta > 0.f ) 
         {
            // First intersection
            intersection1 = ray.origin+ta*ray.direction;
            intersection1.w = 0.f;
            i1 = ( fabs(intersection1.y - cylinder.p0.y) <= cylinder.size.x );
            // Transparency
            if(i1 && materials[cylinder.materialId].textureId != NO_TEXTURE ) 
            {
               float4 color = sphereMapping(cylinder, materials, textures, intersection1 );
               i1 = ((color.x+color.y+color.z) >= sceneInfo.transparentColor ); 
            }
         }

         if( tb > 0.f ) 
         {
            // Second intersection
            intersection2 = ray.origin+tb*ray.direction;
            intersection2.w = 0.f;
            i2 = ( fabs(intersection2.y - cylinder.p0.y) <= cylinder.size.x );
            if(i2 && materials[cylinder.materialId].textureId != NO_TEXTURE ) 
            {
               float4 color = sphereMapping(cylinder, materials, textures, intersection2 );
               i2 = ((color.x+color.y+color.z) >= sceneInfo.transparentColor ); 
            }
         }

         result = i1 || i2;
         if( i1 && i2 )
         {
            float4 O_I1 = intersection1 - ray.origin;
            float4 O_I2 = intersection2 - ray.origin;
            float l1 = vectorLength(O_I1);
            float l2 = vectorLength(O_I2);
            if( l1 < 0.1f ) 
            {
               intersection = intersection2;
            }
            else
            {
               if( l2 < 0.1f )
               {
                  intersection = intersection1;
               }
               else
               {
                  intersection = ( l1<l2 ) ? intersection1 : intersection2;
               }
            }
         }
         else 
         {
            intersection = i1 ? intersection1 : intersection2;
         }
      }
   }

   // Normal to surface
   if( result ) 
   {
      normal   = intersection-cylinder.p0;
      normal.y = 0.f;
      normal.w = 0.f;
      shadowIntensity = 1.f-materials[cylinder.materialId].transparency;
      if( materials[cylinder.materialId].textured ) 
      {
         float4 newCenter;
         newCenter.x = cylinder.p0.x + 5.f*cos(timer*0.58f+intersection.x);
         newCenter.y = cylinder.p0.y + 5.f*sin(timer*0.85f+intersection.y) + intersection.y;
         newCenter.z = cylinder.p0.z + 5.f*sin(cos(timer*1.24f+intersection.z));
         normal = intersection-newCenter;
      }
      normalizeVector( normal );
      result = true;
   }

   /*
   // Soft Shadows
   if( result && computingShadows ) 
   {
      float4 normal = normalToSurface( cylinder, intersection, depth, materials, timer ); // Normal is computed twice!!!
      normalizeVector(ray );
      normalizeVectornormal;
      shadowIntensity = 5.f*fabs(dotProduct(-ray ,normal));
      shadowIntensity = (shadowIntensity>1.f) ? 1.f : shadowIntensity;
   } 
   */
   return result;
}

/*
________________________________________________________________________________

Checkboard intersection
________________________________________________________________________________
*/
__device__ bool planeIntersection( 
   Primitive& primitive,
   Material* materials,
   char*     textures,
   int*      levels,
   Ray       ray, 
   bool      reverse,
   float&    shadowIntensity,
   float4&   intersection,
   float4&   normal,
   float     transparentColor,
   float     timer)
{ 
   bool collision = false;

   float reverted = reverse ? -1.f : 1.f;
   switch( primitive.type ) 
   {
   case ptMagicCarpet:
   case ptCheckboard:
      {
         intersection.y = primitive.p0.y;
         float y = ray.origin.y-primitive.p0.y;
         if( reverted*ray.direction.y<0.f && reverted*ray.origin.y>reverted*primitive.p0.y) 
         {
            normal.x =  0.f;
            normal.y =  1.f;
            normal.z =  0.f;
            intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
            intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
            collision = 
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.z - primitive.p0.z) < primitive.size.y;
         }
         break;
      }
   case ptXZPlane:
      {
         float y = ray.origin.y-primitive.p0.y;
         if( reverted*ray.direction.y<0.f && reverted*ray.origin.y>reverted*primitive.p0.y) 
         {
            normal.x =  0.f;
            normal.y =  1.f;
            normal.z =  0.f;
            intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
            intersection.y = primitive.p0.y;
            intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
            collision = 
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.z - primitive.p0.z) < primitive.size.y;
         }
         if( !collision && reverted*ray.direction.y>0.f && reverted*ray.origin.y<reverted*primitive.p0.y) 
         {
            normal.x =  0.f;
            normal.y = -1.f;
            normal.z =  0.f;
            intersection.x = ray.origin.x+y*ray.direction.x/-ray.direction.y;
            intersection.y = primitive.p0.y;
            intersection.z = ray.origin.z+y*ray.direction.z/-ray.direction.y;
            collision = 
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.z - primitive.p0.z) < primitive.size.y;
         }
         break;
      }
   case ptYZPlane:
      {
         float x = ray.origin.x-primitive.p0.x;
         if( reverted*ray.direction.x<0.f && reverted*ray.origin.x>reverted*primitive.p0.x ) 
         {
            normal.x =  1.f;
            normal.y =  0.f;
            normal.z =  0.f;
            intersection.x = primitive.p0.x;
            intersection.y = ray.origin.y+x*ray.direction.y/-ray.direction.x;
            intersection.z = ray.origin.z+x*ray.direction.z/-ray.direction.x;
            collision = 
               fabs(intersection.y - primitive.p0.y) < primitive.size.y &&
               fabs(intersection.z - primitive.p0.z) < primitive.size.x;
         }
         if( !collision && reverted*ray.direction.x>0.f && reverted*ray.origin.x<reverted*primitive.p0.x ) 
         {
            normal.x = -1.f;
            normal.y =  0.f;
            normal.z =  0.f;
            intersection.x = primitive.p0.x;
            intersection.y = ray.origin.y+x*ray.direction.y/-ray.direction.x;
            intersection.z = ray.origin.z+x*ray.direction.z/-ray.direction.x;
            collision = 
               fabs(intersection.y - primitive.p0.y) < primitive.size.y &&
               fabs(intersection.z - primitive.p0.z) < primitive.size.x;
         }
         break;
      }
   case ptXYPlane:
      {
         float z = ray.origin.z-primitive.p0.z;
         if( reverted*ray.direction.z<0.f && reverted*ray.origin.z>reverted*primitive.p0.z) 
         {
            normal.x =  0.f;
            normal.y =  0.f;
            normal.z =  1.f;
            intersection.z = primitive.p0.z;
            intersection.x = ray.origin.x+z*ray.direction.x/-ray.direction.z;
            intersection.y = ray.origin.y+z*ray.direction.y/-ray.direction.z;
            collision = 
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.y - primitive.p0.y) < primitive.size.y;
         }
         if( !collision && reverted*ray.direction.z>0.f && reverted*ray.origin.z<reverted*primitive.p0.z )
         {
            normal.x =  0.f;
            normal.y =  0.f;
            normal.z = -1.f;
            intersection.z = primitive.p0.z;
            intersection.x = ray.origin.x+z*ray.direction.x/-ray.direction.z;
            intersection.y = ray.origin.y+z*ray.direction.y/-ray.direction.z;
            collision = 
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.y - primitive.p0.y) < primitive.size.y;
         }
         break;
      }
   case ptCamera:
      {
         if( reverted*ray.direction.z<0.f && reverted*ray.origin.z>reverted*primitive.p0.z )
         {
            normal.x =  0.f;
            normal.y =  0.f;
            normal.z =  1.f;
            intersection.z = primitive.p0.z;
            float z = ray.origin.z-primitive.p0.z;
            intersection.x = ray.origin.x+z*ray.direction.x/-ray.direction.z;
            intersection.y = ray.origin.y+z*ray.direction.y/-ray.direction.z;
            collision =
               fabs(intersection.x - primitive.p0.x) < primitive.size.x &&
               fabs(intersection.y - primitive.p0.y) < primitive.size.y;
         }
         break;
      }
   }

   if( collision ) 
   {
      shadowIntensity = 1.f;
      float4 color;
      color = materials[primitive.materialId].color;
      if(materials[primitive.materialId].textureId != NO_TEXTURE)
      {
         color = cubeMapping(primitive, materials, textures, intersection );
      }

      if( materials[primitive.materialId].transparency != 0.f && ((color.x+color.y+color.z)/3.f) >= transparentColor) 
      {
         collision = false;
      }
      else 
      {
         shadowIntensity = ((color.x+color.y+color.z)/3.f*(1.f-materials[primitive.materialId].transparency));
      }
   }
   return collision;
}

#if 0
/*
________________________________________________________________________________

Triangle intersection
________________________________________________________________________________
*/
__device__ bool triangleIntersection( 
   Primitive& triangle, 
   Ray        ray,
   float      timer,
   float4&    intersection,
   float4&    normal,
   bool       computingShadows,
   float&     shadowIntensity,
   float      transparentColor
   ) 
{
   bool result = false;

   float lD = -triangle.p0.x*(triangle.p1.y*triangle.p2.z - triangle.p2.y*triangle.p1.z)
      -triangle.p1.x*(triangle.p2.y*triangle.p0.z - triangle.p0.y*triangle.p2.z)
      -triangle.p2.x*(triangle.p0.y*triangle.p1.z - triangle.p1.y*triangle.p0.z);

   float d = triangle.normal.x*ray.direction.x + triangle.normal.y*ray.direction.y + triangle.normal.z*ray.direction.z;

   d += (d==0.f) ? 0.01f : 0.f;

   float t = -(triangle.normal.x*ray.origin.x + triangle.normal.y*ray.origin.y + triangle.normal.z*ray.origin.z + lD) / d;

   if(t > 0.f)// Triangle in front of the ray
   {
      float4 i = ray.origin+t*ray.direction;

      // 1st side
      float4 I = i - triangle.p0;
      if (dotProduct(triangle.v0,I) <= 0.f)
      {
         // 1st side OK
         I = i - triangle.p1;
         if (dotProduct(triangle.v1,I) <= 0.f)
         {
            // 2nd side OK
            I = i - triangle.p2;
            if (dotProduct(triangle.v2,I) <= 0.f)
            {
               // 1st side OK
               intersection = i;
               normal = triangle.normal;
               result = true;
            }
         }
      }
   }
   return result;
}
#endif // 0

/*
________________________________________________________________________________

Intersection Shader
________________________________________________________________________________
*/
__device__ float4 intersectionShader( 
   SceneInfo& sceneInfo,
   Primitive& primitive, 
   Material*  materials,
   char*      textures,
#ifdef USE_KINECT
   char*      kinectVideo,
#endif // USE_KINECT
   int*       levels,
   float4     intersection,
   float      timer, 
   bool       back )
{
   float4 colorAtIntersection = materials[primitive.materialId].color;
   switch( primitive.type ) 
   {
   case ptEnvironment:
   case ptSphere:
      {
         if(materials[primitive.materialId].textureId != NO_TEXTURE)
         {
            colorAtIntersection = sphereMapping(primitive, materials, textures, intersection );
         }
         break;
      }
   case ptCheckboard :
      {
         if( materials[primitive.materialId].textureId != NO_TEXTURE ) 
         {
            colorAtIntersection = cubeMapping( primitive, materials, textures, intersection );
         }
         else 
         {
            int x = sceneInfo.viewDistance + ((intersection.x - primitive.p0.x)/primitive.p0.w*primitive.materialInfo.x);
            int z = sceneInfo.viewDistance + ((intersection.z - primitive.p0.z)/primitive.p0.w*primitive.materialInfo.y);
            if(x%2==0) 
            {
               if (z%2==0) 
               {
                  colorAtIntersection.x = 1.f-colorAtIntersection.x;
                  colorAtIntersection.y = 1.f-colorAtIntersection.y;
                  colorAtIntersection.z = 1.f-colorAtIntersection.z;
               }
            }
            else 
            {
               if (z%2!=0) 
               {
                  colorAtIntersection.x = 1.f-colorAtIntersection.x;
                  colorAtIntersection.y = 1.f-colorAtIntersection.y;
                  colorAtIntersection.z = 1.f-colorAtIntersection.z;
               }
            }
         }
         break;
      }
   case ptCylinder:
      {
         if(materials[primitive.materialId].textureId != NO_TEXTURE)
         {
            colorAtIntersection = sphereMapping(primitive, materials, textures, intersection );
            //colorAtIntersection = magicCylinderMapping(primitive, materials, textures, intersection, levels, timer);
         }
         break;
      }
#if 0
   case ptTriangle:
      break;
   case ptMagicCarpet:
      {
         if( materials[primitive.materialId].textureId != NO_TEXTURE ) 
         {
            colorAtIntersection = magicCarpetMapping( primitive, materials, textures, intersection, levels, timer );
         }
         break;
      }
   case ptXYPlane:
   case ptYZPlane:
   case ptXZPlane:
      {
         if( materials[primitive.materialId].textureId != NO_TEXTURE ) 
         {
            colorAtIntersection = cubeMapping( primitive, materials, textures, intersection );
         }
         break;
      }
#endif // 0
#ifdef USE_KINECT
   case ptCamera:
      {
         int x = (intersection.x-primitive.p0.x+primitive.size.x)*primitive.materialInfo.x;
         int y = gKinectVideoHeight - (intersection.y-primitive.p0.y+primitive.size.y)*primitive.materialInfo.y;

         x = (x+gKinectVideoWidth)%gKinectVideoWidth;
         y = (y+gKinectVideoHeight)%gKinectVideoHeight;

         if( x>=0 && x<gKinectVideoWidth && y>=0 && y<gKinectVideoHeight ) 
         {
            int index = (y*gKinectVideoWidth+x)*gKinectVideo;
            unsigned char r = kinectVideo[index+2];
            unsigned char g = kinectVideo[index+1];
            unsigned char b = kinectVideo[index+0];
            colorAtIntersection.x = r/256.f;
            colorAtIntersection.y = g/256.f;
            colorAtIntersection.z = b/256.f;
         }
         break;
      }
#endif // USE_KINECT
   }
   return colorAtIntersection;
}

/*
________________________________________________________________________________

Shadows computation
We do not consider the object from which the ray is launched...
This object cannot shadow itself !

We now have to find the intersection between the considered object and the ray 
which origin is the considered 3D float4 and which direction is defined by the 
light source center.
.
. * Lamp                     Ray = Origin -> Light Source Center
.  \
.   \##
.   #### object
.    ##
.      \
.       \  Origin
.--------O-------
.
@return 1.f when pixel is in the shades

________________________________________________________________________________
*/
__device__ float processShadows(
   SceneInfo& sceneInfo,
   BoundingBox* boudingBoxes, int nbActiveBoxes,
   Primitive* primitives,
   Material*  materials,
   char*      textures,
   int*       levels,
   int        nbPrimitives, 
   float4     lampCenter, 
   float4     origin, 
   int        objectId, 
   float      timer)
{
   float result = 0.f;
   int cptBoxes = 0;
   while( result<=sceneInfo.shadowIntensity && cptBoxes < nbActiveBoxes )
   {
      Ray ray;
      ray.origin    = origin;
      ray.direction = lampCenter-origin;
      if(boxIntersection(boudingBoxes[cptBoxes], ray, 0.f, sceneInfo.viewDistance))
      {
         BoundingBox box = boudingBoxes[cptBoxes];
         int cptPrimitives = 0;
         while( result<sceneInfo.shadowIntensity && cptPrimitives<box.nbPrimitives)
         {
            float4 intersection = {0.f,0.f,0.f,0.f};
            float4 normal       = {0.f,0.f,0.f,0.f};
            float  shadowIntensity = 0.f;

            Primitive primitive = primitives[box.primitiveIndex[cptPrimitives]];
            bool hit = false;
            switch(primitive.type)
            {
            case ptEnvironment :
            case ptSphere      : 
               hit = sphereIntersection  ( sceneInfo, primitive, materials, textures, ray, timer, intersection, normal, shadowIntensity ); 
               break;
            case ptCylinder: 
               hit = cylinderIntersection( sceneInfo, primitive, materials, textures, levels, ray, timer, intersection, normal, shadowIntensity ); 
               break;
#if 0
            case ptTriangle: 
               hit = triangleIntersection( primitive, ray, timer, intersection, normal, true, shadowIntensity, sceneInfo.transparentColor ); 
               break;
#endif // 0
            default:
               hit = planeIntersection( primitive, materials, textures, levels, ray, true, shadowIntensity, intersection, normal, sceneInfo.transparentColor, timer ); 
               if( hit ) 
               {
                  float4 O_I = intersection-origin;
                  hit = ( vectorLength(O_I)<vectorLength(ray.direction) );
               }
               break;
            }
            result = hit ? sceneInfo.shadowIntensity : 0.f;
            cptPrimitives++;
         }
      }
      cptBoxes++;
   }
   return (result>1.f) ? 1.f : result;
}

/*
________________________________________________________________________________

Primitive shader
________________________________________________________________________________
*/
__device__ float4 primitiveShader(
   SceneInfo&   sceneInfo,
   BoundingBox* boundingBoxes,
   int          nbActiveBoxes,
   Primitive* primitives,
   int        nbActivePrimitives,
   Lamp*      lamps, 
   int		  nbActiveLamps,
   Material*  materials,
   char*      textures,
#ifdef USE_KINECT
   char*      kinectVideo,
#endif // USE_KINECT
   int*       levels,
   float4     origin,
   float4     normal, 
   int        objectId, 
   float4     intersection, 
   float      timer,
   float4&    refractionFromColor,
   float&     shadowIntensity,
   float4&    totalBlinn)
{
   float4 color = materials[primitives[objectId].materialId].color * materials[primitives[objectId].materialId].specular.z;
   float4 lampsColor = { 0.f, 0.f, 0.f, 0.f };

   // Lamp Impact
   float lambert      = 0.f;
   float totalLambert = 0.f;
   shadowIntensity    = 0.f;

   if( primitives[objectId].type == ptEnvironment )
   {
      totalLambert = 1.f;
      // Final color
      color = intersectionShader( 
         sceneInfo, primitives[objectId], materials, textures, 
#ifdef USE_KINECT
         kinectVideo, 
#endif // USE_KINECT
         levels, intersection, timer, false );
   }
   else 
   {
      for( int cptLamps=0; cptLamps<nbActiveLamps; cptLamps++ ) 
      {
         if( sceneInfo.shadowsEnabled ) 
         {
            shadowIntensity = processShadows(
               sceneInfo, boundingBoxes, nbActiveBoxes,
               primitives, materials, textures, levels, 
               nbActivePrimitives, lamps[cptLamps].center, 
               intersection, objectId, timer );
         }

         float4 lightRay = lamps[cptLamps].center - intersection;
         normalizeVector(lightRay);
         // Lighted object, not in the shades
         lampsColor += lamps[cptLamps].color*lamps[cptLamps].intensity;

         // --------------------------------------------------------------------------------
         // Lambert
         // --------------------------------------------------------------------------------
         lambert = dotProduct(lightRay, normal);
         lambert = (lambert<0.f) ? 0.f : lambert;
         lambert *= (materials[primitives[objectId].materialId].refraction == 0.f) ? lamps[cptLamps].intensity : 1.f;
         lambert *= (1.f-shadowIntensity);
         totalLambert += lambert;

         if( shadowIntensity < sceneInfo.shadowIntensity )
         {
            // --------------------------------------------------------------------------------
            // Blinn - Phong
            // --------------------------------------------------------------------------------
            float4 viewRay = intersection - origin;
            normalizeVector(viewRay);

            float4 blinnDir = lightRay - viewRay;
            float temp = sqrt(dotProduct(blinnDir,blinnDir));
            if (temp != 0.f ) 
            {
               // Specular reflection
               blinnDir = (1.f / temp) * blinnDir;

               float blinnTerm = dotProduct(blinnDir,normal);
               blinnTerm = ( blinnTerm < 0.f) ? 0.f : blinnTerm;

               blinnTerm = materials[primitives[objectId].materialId].specular.x * pow(blinnTerm,materials[primitives[objectId].materialId].specular.y);
               totalBlinn += lamps[cptLamps].color * lamps[cptLamps].intensity * blinnTerm;
            }
         }
      }
      // Final color
      float4 intersectionColor = intersectionShader( 
         sceneInfo, primitives[objectId], materials, textures, 
#ifdef USE_KINECT
         kinectVideo, 
#endif // USE_KINECT
         levels, intersection, timer, false );

      color += totalLambert*intersectionColor*lampsColor;
      saturateVector(color);

      refractionFromColor = intersectionColor; // Refraction depending on color;
      saturateVector( totalBlinn );
   }

   return color;
}

/*
________________________________________________________________________________

Intersections with lamps
________________________________________________________________________________
*/
__device__ bool intersectionWithLamps( 
   Lamp* lamps, 
   int nbActiveLamps,
   Ray ray, 
   float4& lampColor)
{
   int intersections = 0; 
   lampColor.x = 0.f;
   lampColor.y = 0.f;
   lampColor.z = 0.f;

   for( int cptLamps = 0; cptLamps<nbActiveLamps; cptLamps++ ) 
   {
      float4 O_C = ray.origin - lamps[cptLamps].center; 
      float4 intersection;

      Ray r; // To do
      r.origin = ray.origin;
      r.direction = ray.direction - ray.origin;
      if( lampIntersection( lamps[cptLamps], r, O_C, intersection ) ) 
      {
         intersections++;
         lampColor += lamps[cptLamps].color;
      }
   }
   if( intersections != 0 ) 
   {
      lampColor /= intersections;
   }
   return (intersections != 0 );
}

/*
________________________________________________________________________________

Intersections with primitives
________________________________________________________________________________
*/
__device__ bool intersectionWithPrimitives(
   SceneInfo& sceneInfo,
   BoundingBox* boundingBoxes, int nbActiveBoxes,
   Primitive* primitives, int nbActivePrimitives,
   Material* materials, char* textures, int* levels,
   Ray     ray, 
   float   timer, 
   int&    closestPrimitive, 
   float4& closestIntersection,
   float4& closestNormal)
{
   bool intersections = false; 
   float minDistance  = sceneInfo.viewDistance;
   Ray r;
   r.origin    = ray.origin;
   r.direction = ray.direction - ray.origin;

   float4 intersection = {0.f,0.f,0.f,0.f};
   float4 normal       = {0.f,0.f,0.f,0.f};

   for( int cptBoxes = 0; cptBoxes < nbActiveBoxes; ++cptBoxes )
   {
      BoundingBox& box = boundingBoxes[cptBoxes];
      if( boxIntersection(box, r, 0.f, sceneInfo.viewDistance) )
      {
         if( sceneInfo.renderBoxes ) 
         {
            closestPrimitive = cptBoxes;
            return true;
         }
         int cptObjects = 0;
         //bool stop = false;
         while( /*!stop &&*/ cptObjects<box.nbPrimitives)
         { 
            bool i = false;
            float shadowIntensity = 0.f;
            Primitive& primitive = primitives[box.primitiveIndex[cptObjects]];

            float distance = vectorLength( ray.origin - primitive.p0 ) - primitive.size.x; // TODO! Not sure if i should keep it
            if( distance < minDistance )
            {
               switch( primitive.type )
               {
               case ptEnvironment :
               case ptSphere      : 
                  i = sphereIntersection  ( sceneInfo, primitive, materials, textures, r, timer, intersection, normal, shadowIntensity ); 
                  break;
               case ptCylinder: 
                  i = cylinderIntersection( sceneInfo, primitive, materials, textures, levels, r, timer, intersection, normal, shadowIntensity); 
                  break;
#if 0
               case ptTriangle: 
                  i = triangleIntersection( primitive, r, timer, intersection, normal, false, shadowIntensity, transparentColor ); 
                  break;
#endif // 0
               default        : 
                  i = planeIntersection   ( primitive, materials, textures, levels, r, false, shadowIntensity, intersection, normal, sceneInfo.transparentColor, timer); 
                  break;
               }

               if( i ) 
               {
                  float distance = vectorLength( ray.origin - intersection );
                  //stop = (cptObjects==0 && distance>minDistance);
                  if(distance>1.f && distance<minDistance) 
                  {
                     // Only keep intersection with the closest object
                     minDistance         = distance;
                     closestPrimitive    = cptObjects;
                     closestIntersection = intersection;
                     closestNormal       = normal;
                     intersections       = true;
                  } 
               }
            }
            cptObjects++;
         }
      }
   }
   return intersections;
}

/*
________________________________________________________________________________

Calculate the reflected vector                   
                                                  
                  ^ Normal to object surface (N)  
Reflection (O_R)  |                              
                \ |  Eye (O_E)                    
                 \| /                             
  ----------------O--------------- Object surface 
        closestIntersection                      
                                                   
============================================================================== 
colours                                                                                    
------------------------------------------------------------------------------ 
We now have to know the colour of this intersection                                        
Color_from_object will compute the amount of light received by the
intersection float4 and  will also compute the shadows. 
The resulted color is stored in result.                     
The first parameter is the closest object to the intersection (following 
the ray). It can  be considered as a light source if its inner light rate 
is > 0.                            
________________________________________________________________________________
*/
__device__ float4 launchRay( 
   BoundingBox* boundingBoxes, int nbActiveBoxes,
   Primitive* primitives, int nbActivePrimitives,
   Lamp*      lamps, int nbActiveLamps,
   Material*  materials,
   char* textures,
#ifdef USE_KINECT
   char*      kinectVideo, 
#endif // USE_KINECT
   int* levels,
   Ray        ray, 
   float      timer, 
   SceneInfo& sceneInfo,
   float4&    intersection,
   float&     depthOfField)
{
   float4 intersectionColor   = {0.f,0.f,0.f,0.f};
   float4 closestIntersection = {0.f,0.f,0.f,0.f};
   float4 firstIntersection   = {0.f,0.f,0.f,0.f};
   float4 normal              = {0.f,0.f,0.f,0.f};
   int    closestPrimitive;
   bool   carryon           = true;
   Ray    rayOrigin         = ray;
   float  initialRefraction = 1.0f;
   int    iteration         = 0;
   Ray    O_R = ray;
   float4 O_E;
   float4 recursiveColor[gNbIterations+1];
   float4 recursiveRatio[gNbIterations+1];
   float4 recursiveBlinn[gNbIterations+1];

   memset(recursiveColor,0,sizeof(float4)*(sceneInfo.nbRayIterations+1));
   memset(recursiveRatio,0,sizeof(float4)*(sceneInfo.nbRayIterations+1));
   memset(recursiveBlinn,0,sizeof(float4)*(sceneInfo.nbRayIterations+1));

   recursiveColor[0] = sceneInfo.backgroundColor;

   // Variable declarations
   float  shadowIntensity = 0.f;
   float4 refractionFromColor;
   float4 reflectedTarget;

   while( iteration<sceneInfo.nbRayIterations && carryon ) 
   {
      // Compute intesection with lamps
      if( intersectionWithLamps( lamps, nbActiveLamps, rayOrigin, intersectionColor ) )
      {
         recursiveColor[iteration] = intersectionColor;
         carryon = false;
      }
      else
      {
         carryon = true;
      }

      // If no intersection with lamps detected. Now compute intersection with Primitives
      if( carryon ) 
      {
         carryon = intersectionWithPrimitives(
            sceneInfo,
            boundingBoxes, nbActiveBoxes,
            primitives, nbActivePrimitives,
            materials, textures, levels,
            rayOrigin,
            timer, 
            closestPrimitive, closestIntersection, 
            normal);
      }

      if( carryon ) 
      {
         if( sceneInfo.renderBoxes ) 
         {
            recursiveColor[iteration] = materials[closestPrimitive%10].color;
         }
         else 
         {
            if ( iteration==0 )
            {
               firstIntersection = closestIntersection;
            }

            // Get object color
            recursiveColor[iteration] = primitiveShader( 
               sceneInfo,
               boundingBoxes, nbActiveBoxes,
               primitives, nbActivePrimitives, lamps, nbActiveLamps, materials, textures, 
   #ifdef USE_KINECT
               kinectVideo, 
   #endif // USE_KINECT
               levels,
               rayOrigin.origin, normal, closestPrimitive, closestIntersection, 
               timer, refractionFromColor, shadowIntensity, recursiveBlinn[iteration] );

            if( shadowIntensity != 1.f ) // No reflection/refraction if in shades
            {
               // ----------
               // Refraction
               // ----------
               if( materials[primitives[closestPrimitive].materialId].transparency != 0.f ) 
               {
                  // ----------
                  // Refraction
                  // ----------
                  // Replace the normal using the intersection color
                  // r,g,b become x,y,z... What the fuck!!
                  if( materials[primitives[closestPrimitive].materialId].textureId != NO_TEXTURE) 
                  {
                     refractionFromColor -= 0.5f;
                     normal *= refractionFromColor;
                  }

                  O_E = rayOrigin.origin - closestIntersection;
                  float refraction = materials[primitives[closestPrimitive].materialId].refraction;
                  refraction = (refraction == initialRefraction) ? 1.0f : refraction;
                  vectorRefraction( O_R.direction, O_E, refraction, normal, initialRefraction );
                  reflectedTarget = closestIntersection - O_R.direction;
                  initialRefraction = refraction;

                  recursiveRatio[iteration].x = materials[primitives[closestPrimitive].materialId].transparency;
                  recursiveRatio[iteration].z = 1.f;
               }
               else 
               {
                  // ----------
                  // Reflection
                  // ----------
                  if( materials[primitives[closestPrimitive].materialId].reflection != 0.f ) 
                  {
                     O_E = rayOrigin.origin - closestIntersection;
                     vectorReflection( O_R.direction, O_E, normal );

                     reflectedTarget = closestIntersection - O_R.direction;

                     recursiveRatio[iteration].x = materials[primitives[closestPrimitive].materialId].reflection;
                  }
                  else 
                  {
                     carryon = false;
                  }         
               }
            }
            else 
            {
               carryon = false;
            }
            rayOrigin.origin    = closestIntersection; 
            rayOrigin.direction = reflectedTarget;
         }

         iteration++; 
      }
   }

   for( int i=iteration-1; i>=0; --i ) 
   {
      recursiveColor[i] = recursiveColor[i+1]*recursiveRatio[i].x + recursiveColor[i]*(1.f-recursiveRatio[i].x);
      recursiveColor[i] += recursiveBlinn[i];
   }
   intersectionColor = recursiveColor[0];

   // Specular reflection
   //intersectionColor += recursiveBlinn[0];

   saturateVector( intersectionColor );
   intersection = closestIntersection;

   float4 O_I = firstIntersection - ray.origin;
#if 1
   // --------------------------------------------------
   // Attenation effect (Fog)
   // --------------------------------------------------
   float len = 1.f-(vectorLength(O_I)/sceneInfo.viewDistance);
   len = (len>0.f) ? len : 0.f; 
   intersectionColor.x = intersectionColor.x * len;
   intersectionColor.y = intersectionColor.y * len;
   intersectionColor.z = intersectionColor.z * len;
#endif // 0

   // Depth of field
   float4 FI_I = firstIntersection - ray.direction;
   depthOfField = (vectorLength(O_I)-depthOfField)/sceneInfo.viewDistance;
   return intersectionColor;
}


/*
________________________________________________________________________________

Main Kernel!!!
________________________________________________________________________________
*/
__global__ void k_raytracingRenderer(
   BoundingBox* BoundingBoxes, int nbActiveBoxes,
   Primitive*   primitives,    int nbActivePrimitives,
   Lamp*        lamps,         int nbActiveLamps,
   Material*    materials,
   char*        textures,
#ifdef USE_KINECT
   char*        kinectVideo,
#endif // USE_KINECT
   int*         levels,
   Ray          ray,
   float4       angles,
   SceneInfo    sceneInfo,
   float        timer,
   DepthOfFieldInfo depthOfField,
   float4*      postProcessingBuffer)
{
   int x = blockDim.x*blockIdx.x + threadIdx.x;
   int y = blockDim.y*blockIdx.y + threadIdx.y;
   int index = y*(sceneInfo.width/sceneInfo.draft)+x;

   float4 rotationCenter = {0.f,0.f,0.f,0.f};

   postProcessingBuffer[index].x = 0.f;
   postProcessingBuffer[index].y = 0.f;
   postProcessingBuffer[index].z = 0.f;
   postProcessingBuffer[index].w = 0.f;
   float dof = depthOfField.pointOfFocus;
   float4 intersection;

   if( sceneInfo.supportFor3DVision )
   {
      int halfWidth  = sceneInfo.width/2;

      Ray eyeRay;
      if( x<halfWidth ) 
      {
         // Left eye
         eyeRay.origin.x = ray.origin.x + sceneInfo.width3DVision;
         eyeRay.origin.y = ray.origin.y;
         eyeRay.origin.z = ray.origin.z;

         eyeRay.direction.x = ray.direction.x - 8.f*(float)(x - (sceneInfo.width/2) + halfWidth/2 );
         eyeRay.direction.y = ray.direction.y + 8.f*(float)(y - (sceneInfo.height/2));
         eyeRay.direction.z = ray.direction.z;
      }
      else
      {
         // Right eye
         eyeRay.origin.x = ray.origin.x - sceneInfo.width3DVision;
         eyeRay.origin.y = ray.origin.y;
         eyeRay.origin.z = ray.origin.z;

         eyeRay.direction.x = ray.direction.x - 8.f*(float)(x - (sceneInfo.width/2) - halfWidth/2);
         eyeRay.direction.y = ray.direction.y + 8.f*(float)(y - (sceneInfo.height/2));
         eyeRay.direction.z = ray.direction.z;
      }
      
      vectorRotation( eyeRay.origin, rotationCenter, angles );
      vectorRotation( eyeRay.direction, rotationCenter, angles );

      // Lamp is always behind viewer
      lamps[0].center.x = 4.f*ray.origin.x;
      lamps[0].center.y = 4.f*ray.origin.y;
      lamps[0].center.z = 2.f*ray.origin.z;

      postProcessingBuffer[index] = launchRay(
         BoundingBoxes, nbActiveBoxes,
         primitives, nbActivePrimitives,
         lamps, nbActiveLamps,
         materials,
         textures, 
   #ifdef USE_KINECT
         kinectVideo, 
   #endif // USE_KINECT
         levels,
         eyeRay, timer, 
         sceneInfo,
         intersection,
         dof);
      postProcessingBuffer[index].w = dof;
   }
   else
   {
      ray.direction.x = ray.direction.x - 8.f*(float)(sceneInfo.draft*x - (sceneInfo.width/2));
      ray.direction.y = ray.direction.y + 8.f*(float)(sceneInfo.draft*y - (sceneInfo.height/2));
      vectorRotation( ray.origin, rotationCenter, angles );
      vectorRotation( ray.direction, rotationCenter, angles );

      // Lamp is always behind viewer
      lamps[0].center.x = 4.f*ray.origin.x;
      lamps[0].center.y = 4.f*ray.origin.y;
      lamps[0].center.z = 2.f*ray.origin.z;

      postProcessingBuffer[index] = launchRay(
         BoundingBoxes, nbActiveBoxes,
         primitives, nbActivePrimitives,
         lamps, nbActiveLamps,
         materials,
         textures, 
   #ifdef USE_KINECT
         kinectVideo, 
   #endif // USE_KINECT
         levels,
         ray, timer, 
         sceneInfo,
         intersection,
         dof);
   
      postProcessingBuffer[index].w = dof;
   }
}

/*
________________________________________________________________________________

Post processing effects
________________________________________________________________________________
*/
__global__ void k_postProcessingEffects(
   SceneInfo        sceneInfo,
   DepthOfFieldInfo depthOfFieldInfo,
   float4*          postProcessingBuffer,
   float*           randoms,
   char*            bitmap) 
{
   int x = blockDim.x*blockIdx.x + threadIdx.x;
   int y = blockDim.y*blockIdx.y + threadIdx.y;
   int index = y*(sceneInfo.width/sceneInfo.draft)+x;
   if( depthOfFieldInfo.enabled )
   {
      float  depth = depthOfFieldInfo.strength*postProcessingBuffer[index].w;
      int    wh = sceneInfo.width*sceneInfo.height;

      float4 localColor;
      localColor.x = 0.f;
      localColor.y = 0.f;
      localColor.z = 0.f;

      for( int i=0; i<depthOfFieldInfo.iterations; ++i )
      {
         int ix = i%wh;
         int iy = (i+sceneInfo.width)%wh;
         int xx = x+depth*randoms[ix];
         int yy = y+depth*randoms[iy];
         if( xx>=0 && xx<sceneInfo.width && yy>=0 && yy<sceneInfo.height )
         {
            int localIndex = yy*sceneInfo.width+xx;
            if( localIndex>=0 && localIndex<wh )
            {
               localColor += postProcessingBuffer[localIndex];
            }
         }
      }
      localColor /= depthOfFieldInfo.iterations;
      localColor.w = 0.f;

      makeOpenGLColor( localColor, bitmap, index ); 
   }
   else 
   {
      makeOpenGLColor( postProcessingBuffer[index], bitmap, index ); 
   }
}

/*
________________________________________________________________________________

GPU initialization
________________________________________________________________________________
*/
extern "C" void initialize_scene( 
   int width, int height, int nbPrimitives, int nbLamps, int nbMaterials, int nbTextures, int nbLevels )
{
   // Scene resources
   checkCudaErrors(hipMalloc( (void**)&d_boundingBoxes, NB_MAX_BOXES*sizeof(BoundingBox)));
   checkCudaErrors(hipMalloc( (void**)&d_primitives,    nbPrimitives*sizeof(Primitive)));
   checkCudaErrors(hipMalloc( (void**)&d_lamps,         nbLamps*sizeof(Lamp)));
   checkCudaErrors(hipMalloc( (void**)&d_materials,     nbMaterials*sizeof(Material)));
   checkCudaErrors(hipMalloc( (void**)&d_textures,      nbTextures*gTextureDepth*gTextureWidth*gTextureHeight));
   checkCudaErrors(hipMalloc( (void**)&d_randoms,       width*height*sizeof(float)));
   checkCudaErrors(hipMalloc( (void**)&d_levels,        nbLevels*sizeof(int)));

   // Rendering canvas
   checkCudaErrors(hipMalloc( (void**)&d_postProcessingBuffer,  width*height*sizeof(float4)));
   checkCudaErrors(hipMalloc( (void**)&d_bitmap,                width*height*gColorDepth*sizeof(char)));

#ifdef USE_KINECT
   // Kinect video and depth buffers
   checkCudaErrors(hipMalloc( (void**)&d_kinectVideo,   gKinectVideo*gKinectVideoWidth*gKinectVideoHeight*sizeof(char)));
   checkCudaErrors(hipMalloc( (void**)&d_kinectDepth,   gKinectDepth*gKinectDepthWidth*gKinectDepthHeight*sizeof(char)));
#endif // USE_KINECT
}

/*
________________________________________________________________________________

GPU finalization
________________________________________________________________________________
*/
extern "C" void finalize_scene()
{
   checkCudaErrors(hipFree( d_boundingBoxes ));
   checkCudaErrors(hipFree( d_primitives ));
   checkCudaErrors(hipFree( d_lamps ));
   checkCudaErrors(hipFree( d_materials ));
   checkCudaErrors(hipFree( d_textures ));
   checkCudaErrors(hipFree( d_randoms ));
   checkCudaErrors(hipFree( d_levels ));
   checkCudaErrors(hipFree( d_postProcessingBuffer ));
   checkCudaErrors(hipFree( d_bitmap ));
#ifdef USE_KINECT
   checkCudaErrors(hipFree( d_kinectVideo ));
   checkCudaErrors(hipFree( d_kinectDepth ));
#endif // USE_KINECT
}

/*
________________________________________________________________________________

CPU -> GPU data transfers
________________________________________________________________________________
*/
extern "C" void h2d_scene( 
   BoundingBox* boundingBoxes, int nbActiveBoxes,
   Primitive*  primitives,     int nbPrimitives,
   Lamp*       lamps,          int nbLamps )
{
   checkCudaErrors(hipMemcpy( d_boundingBoxes, boundingBoxes, nbActiveBoxes*sizeof(BoundingBox), hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_primitives,    primitives,    nbPrimitives*sizeof(Primitive),    hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_lamps,         lamps,         nbLamps*sizeof(Lamp),              hipMemcpyHostToDevice ));
}

extern "C" void h2d_materials( 
   Material*  materials, int nbActiveMaterials,
   char*      textures , int nbActiveTextures,
   float*     randoms,   int nbRandoms,
   int*       levels,    int levelSize)
{
   checkCudaErrors(hipMemcpy( d_materials, materials, nbActiveMaterials*sizeof(Material), hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_textures,  textures,  nbActiveTextures*sizeof(char)*gTextureDepth*gTextureWidth*gTextureHeight,  hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_randoms,   randoms,   nbRandoms*sizeof(float), hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_levels,    levels,    levelSize*sizeof(int), hipMemcpyHostToDevice ));
}

#ifdef USE_KINECT
extern "C" void h2d_kinect( 
   char* kinectVideo, int videoSize,
   char* kinectDepth, int depthSize )
{
   checkCudaErrors(hipMemcpy( d_kinectVideo, kinectVideo, videoSize*sizeof(char), hipMemcpyHostToDevice ));
   checkCudaErrors(hipMemcpy( d_kinectDepth, kinectDepth, depthSize*sizeof(char), hipMemcpyHostToDevice ));
}
#endif // USE_KINECT

/*
________________________________________________________________________________

GPU -> CPU data transfers
________________________________________________________________________________
*/
extern "C" void d2h_bitmap( char* bitmap, const SceneInfo sceneInfo )
{
   checkCudaErrors(hipMemcpy( 
      bitmap, 
      d_bitmap, 
      sceneInfo.width*sceneInfo.height*gColorDepth*sizeof(char), 
      hipMemcpyDeviceToHost ));
}

/*
________________________________________________________________________________

Kernel launcher
________________________________________________________________________________
*/
extern "C" void cudaRender(
   dim3 blockSize, int sharedMemSize,
   int nbActiveBoxes, int nbPrimitives, int nbLamps,
   Ray ray, float4 angles,
   SceneInfo sceneInfo,
   DepthOfFieldInfo depthOfFieldInfo,
   float timer)
{
   int2 size;
   size.x = static_cast<int>(sceneInfo.width/sceneInfo.draft);
   size.y = static_cast<int>(sceneInfo.height/sceneInfo.draft);
   dim3 grid((size.x+blockSize.x-1)/blockSize.x,(size.y+blockSize.y-1)/blockSize.y,1);

   k_raytracingRenderer<<<grid,blockSize,sharedMemSize>>>(
      d_boundingBoxes, nbActiveBoxes,
      d_primitives, nbPrimitives, 
      d_lamps, nbLamps,
      d_materials,
      d_textures, 
#ifdef USE_KINECT
      d_kinectVideo, 
#endif // USE_KINECT
      d_levels,
      ray, angles, 
      sceneInfo,
      timer,
      depthOfFieldInfo,
      d_postProcessingBuffer);

   k_postProcessingEffects<<<grid,blockSize>>>(
      sceneInfo, 
      depthOfFieldInfo, 
      d_postProcessingBuffer,
      d_randoms, 
      d_bitmap );
}
