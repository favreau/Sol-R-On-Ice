#include "hip/hip_runtime.h"
/* 
* OpenCL Raytracer
* Copyright (C) 2011-2012 Cyrille Favreau <cyrille_favreau@hotmail.com>
*
* This library is free software; you can redistribute it and/or
* modify it under the terms of the GNU Library General Public
* License as published by the Free Software Foundation; either
* version 2 of the License, or (at your option) any later version.
*
* This library is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
* Library General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

/*
* Author: Cyrille Favreau <cyrille_favreau@hotmail.com>
*
*/

#include <cutil_inline.h>
#include <cutil_math.h>
#include "CudaDataTypes.h"

#include "Scene.cuh"
#include "Vectors.cuh"
#include "Intersections.cuh"
#include "Colors.cuh"

// Cuda Runtime paramters

#define BLOCK_SIZE 16

#if 0
// Not used so far
const int gDepthOfFieldComplexity = 1;
const int gRayCastingIterations   = 1;
const float4 gAmbientLight = { 0.4f, 0.4f, 0.4f, 0.f };
#endif // 0

// Device arrays
__constant__ __device__ Primitive* d_primitives; 
__constant__ __device__ Lamp*      d_lamps;
__constant__ __device__ Material*  d_materials;
__constant__ __device__ char*      d_textures;
__constant__ __device__ float*     d_randoms;
__constant__ __device__ int*       d_levels;
__device__   float4*    d_depthOfField;
__device__   char*      d_bitmap;

__constant__ __device__ char*      d_kinectVideo;
__constant__ __device__ char*      d_kinectDepth;

/*
* Shadows computation
* We do not consider the object from which the ray is launched...
* This object cannot shadow itself !
* 
* We now have to find the intersection between the considered object and the ray which origin is the considered 3D float4
* and which direction is defined by the light source center.
* 
* * Lamp                     Ray = Origin -> Light Source Center
*   \
*    \##
*    #### object
*     ##
*       \
*        \  Origin
* --------O-------
* 
* @return 1.f when pixel is in the shades
*/
__device__ float shadow(
   Primitive* primitives,
   Material*  materials,
   char*      textures,
   int*       levels,
   int        nbPrimitives, 
   float4     lampCenter, 
   float4     origin, 
   int        objectId, 
   float      timer,
   float      transparentColor)
{
   float4 lookAround[7] = 
   { 
      {   0.f,   0.f,   0.f,  0.f },
      { -50.f,   0.f,   0.f,  0.f },
      {   0.f,  50.f,   0.f,  0.f },
      {  50.f,   0.f,   0.f,  0.f },
      {   0.f, -50.f,   0.f,  0.f },
      {   0.f,   0.f,   1.f,  0.f },
      {   0.f,   0.f,  -1.f,  0.f } 
   };

   float result = 0.f;
   int   cptPrimitives = 0;
   int   collision = 0;
   float localShadow = 0.f;
   while( result<1.f && (collision<gNbMaxShadowCollisions) && (cptPrimitives<nbPrimitives) ) 
   {
      float shadowIntensity = 0.f;
      int hitNb = 0;
      float4 intersection = {0.f,0.f,0.f,0.f};
      float4 normal       = {0.f,0.f,0.f,0.f};
      for( int la=0; la<gSoftShadowPrecision; ++la )
      {
         float4 O_L = (lampCenter+lookAround[la])-origin;
         bool hit = false;

         switch(primitives[cptPrimitives].type)
         {
         case ptSphere  : hit = sphereIntersection  ( primitives[cptPrimitives], materials, textures, origin, O_L, timer, intersection, normal, shadowIntensity, transparentColor ); break;
         case ptCylinder: hit = cylinderIntersection( primitives[cptPrimitives], materials, textures, levels, origin, O_L, timer, intersection, normal, shadowIntensity, transparentColor ); break;
         case ptTriangle: hit = triangleIntersection( primitives[cptPrimitives], origin, O_L, timer, intersection, normal, true, shadowIntensity, transparentColor ); break;
         default:
            hit = planeIntersection( primitives[cptPrimitives], materials, textures, levels, origin, O_L, true, shadowIntensity, intersection, normal, transparentColor, timer ); 
            if( hit ) 
            {
               float4 O_I = intersection-origin;
               hit = ( vectorLength(O_I)<vectorLength(O_L) );
            }
            break;
         }

         if( hit ) 
         {
            hitNb++;
            if( primitives[cptPrimitives].type == ptSphere || primitives[cptPrimitives].type == ptCylinder || primitives[cptPrimitives].type == ptTriangle )
            {
               // Shadow exists only if object is between origin and lamp
               float4 O_I = intersection-origin;
               shadowIntensity = (vectorLength(O_I) < vectorLength(O_L)) ? shadowIntensity : 0.f;
            }
            localShadow += shadowIntensity;
         }
      }
      collision += (hitNb == 0) ? 0 : 1;
      cptPrimitives++; 
   }
   result = localShadow/gSoftShadowPrecision;
   return (result>1.f) ? 1.f : result;
}


/*
* colorFromObject 
*/
__device__ float4 colorFromObject(
   Primitive* primitives,
   int        nbActivePrimitives,
   Lamp*      lamps, 
   int		  nbActiveLamps,
   Material*  materials,
   char*      textures,
   char*      kinectVideo,
   int*       levels,
   float4     origin,
   float4     normal, 
   int        objectId, 
   float4     intersection, 
   float      timer,
   float4&    refractionFromColor,
   float&     shadowIntensity,
   float4&    totalBlinn,
   float      transparentColor)
{
   float4 color = materials[primitives[objectId].materialId].color * materials[primitives[objectId].materialId].specular.z;
   float4 lampsColor = { 0.f, 0.f, 0.f, 0.f };

   // Lamp Impact
   float lambert      = 0.f;
   float totalLambert = 0.f;
   shadowIntensity    = 0.f;

   for( int cptLamps=0; cptLamps<nbActiveLamps; cptLamps++ ) 
   {
      shadowIntensity = shadow( primitives, materials, textures, levels, nbActivePrimitives, lamps[cptLamps].center, intersection, objectId, timer, transparentColor );

      //if( (shadowIntensity) != 1.f )
      {
         // Lighted object, not in the shades
         lampsColor += lamps[cptLamps].color*lamps[cptLamps].intensity;

         // --------------------------------------------------------------------------------
         // Lambert
         // --------------------------------------------------------------------------------
         float4 lightRay = lamps[cptLamps].center - intersection;
         normalizeVector(lightRay);
         lambert = dotProduct(lightRay, normal);
         lambert = (lambert<0.f) ? 0.f : lambert;
         lambert *= (materials[primitives[objectId].materialId].refraction == 0.f) ? lamps[cptLamps].intensity : 1.f;
         lambert *= (1.f-shadowIntensity);
         totalLambert += lambert;

         // --------------------------------------------------------------------------------
         // Blinn - Phong
         // --------------------------------------------------------------------------------
         float4 viewRay = intersection - origin;
         normalizeVector(viewRay);

         float4 blinnDir = lightRay - viewRay;
         float temp = sqrt(dotProduct(blinnDir,blinnDir));
         if (temp != 0.f ) 
         {
            // Specular reflection
            blinnDir = (1.f / temp) * blinnDir;

            float blinnTerm = dotProduct(blinnDir,normal);
            blinnTerm = ( blinnTerm < 0.f) ? 0.f : blinnTerm;

            blinnTerm = materials[primitives[objectId].materialId].specular.x * pow(blinnTerm,materials[primitives[objectId].materialId].specular.y);
            totalBlinn += lamps[cptLamps].color * lamps[cptLamps].intensity * blinnTerm;
         }
      }
   }

   // Final color
   float4 intersectionColor = objectColorAtIntersection( primitives[objectId], materials, textures, kinectVideo, levels, intersection, timer, false );

   color += totalLambert*intersectionColor*lampsColor;
   saturateVector(color);

   refractionFromColor = intersectionColor; // Refraction depending on color;
   saturateVector( totalBlinn );
   return color;
}

/**
* ________________________________________________________________________________
* Intersections with Objects
* ________________________________________________________________________________
*/
__device__ bool intersectionWithLamps( 
   Lamp* lamps, int nbActiveLamps,
   float4 origin, float4 target, 
   float4& lampColor)
{
   int intersections = 0; 
   lampColor.x = 0.f;
   lampColor.y = 0.f;
   lampColor.z = 0.f;

   for( int cptLamps = 0; cptLamps<nbActiveLamps; cptLamps++ ) 
   {
      float4 O_C = origin - lamps[cptLamps].center; 
      float4 ray = target - origin;
      float4 intersection;

      if( lampIntersection( lamps[cptLamps], origin, ray, O_C, intersection ) ) 
      {
         intersections++;
         lampColor += lamps[cptLamps].color;
      }
   }
   if( intersections != 0 ) 
   {
      lampColor /= intersections;
   }
   return (intersections != 0 );
}

/**
* ________________________________________________________________________________
* Intersections with Objects
* ________________________________________________________________________________
*/
__device__ bool intersectionWithPrimitives( 
   Primitive* primitives, int nbActivePrimitives,
   Material* materials, char* textures, int* levels,
   float4  origin, float4 target, 
   float   timer, 
   int&    closestPrimitive, 
   float4& closestIntersection,
   float4& closestNormal,
   float   transparentColor)
{
   bool intersections = false; 
   float minDistance  = gMaxViewDistance; 
   float4 ray = target - origin; 
   float4 intersection = {0.f,0.f,0.f,0.f};
   float4 normal       = {0.f,0.f,0.f,0.f};

   for( int cptObjects = 0; cptObjects<nbActivePrimitives; cptObjects++ )
   { 
      bool i = false; 
      float shadowIntensity;

      switch( primitives[cptObjects].type )
      {
      case ptSphere  : i = sphereIntersection  ( primitives[cptObjects], materials, textures, origin, ray, timer, intersection, normal, shadowIntensity, transparentColor ); break;
      case ptCylinder: i = cylinderIntersection( primitives[cptObjects], materials, textures, levels, origin, ray, timer, intersection, normal, shadowIntensity, transparentColor); break;
      case ptTriangle: i = triangleIntersection( primitives[cptObjects], origin, ray, timer, intersection, normal, false, shadowIntensity, transparentColor ); break;
      default        : i = planeIntersection   ( primitives[cptObjects], materials, textures, levels, origin, ray, false, shadowIntensity, intersection, normal, transparentColor, timer); break;
      }

      if( i ) 
      {
         float distance = vectorLength( origin - intersection );

         if(distance>1.f && distance<minDistance) 
         {
            // Only keep intersection with the closest object
            minDistance         = distance;
            closestPrimitive    = cptObjects;
            closestIntersection = intersection;
            closestNormal       = normal;
            intersections       = true;
         } 
      }
   }
   return intersections;
}

/**
*  ------------------------------------------------------------------------------ 
* Ray Intersections
*  ============================================================================== 
*  Calculate the reflected vector                   
*                                                  
*                  ^ Normal to object surface (N)  
* Reflection (O_R)  |                              
*                 \ |  Eye (O_E)                    
*                  \| /                             
*   ----------------O--------------- Object surface 
*          closestIntersection                      
*                                                   
*  ============================================================================== 
*  colours                                                                                    
*  ------------------------------------------------------------------------------ 
*  We now have to know the colour of this intersection                                        
*  Color_from_object will compute the amount of light received by the
*  intersection float4 and  will also compute the shadows. 
*  The resulted color is stored in result.                     
*  The first parameter is the closest object to the intersection (following 
*  the ray). It can  be considered as a light source if its inner light rate 
*  is > 0.                            
*  ------------------------------------------------------------------------------ 
*/
__device__ float4 launchRay( 
   Primitive* primitives, int nbActivePrimitives,
   Lamp*      lamps, int nbActiveLamps,
   Material*  materials, char* textures,
   char*      kinectVideo, int* levels,
   float4     origin, float4 target, 
   float      timer, float transparentColor,
   float4&    intersection,
   float&     depthOfField)
{
   float4 intersectionColor   = {0.f,0.f,0.f,0.f};
   float4 closestIntersection = {0.f,0.f,0.f,0.f};
   float4 firstIntersection   = {0.f,0.f,0.f,0.f};
   float4 normal              = {0.f,0.f,0.f,0.f};
   int    closestPrimitive;
   bool   carryon           = true;
   float4 rayOrigin         = origin;
   float4 rayTarget         = target;
   float  initialRefraction = 1.0f;
   int    iteration         = 0;
   float4 O_R;
   float4 O_E;
   float4 recursiveColor[gNbIterations];
   float4 recursiveRatio[gNbIterations];
   float4 recursiveBlinn[gNbIterations];

   memset(recursiveColor,0,sizeof(float4)*gNbIterations);
   memset(recursiveRatio,0,sizeof(float4)*gNbIterations );
   memset(recursiveBlinn,0,sizeof(float4)*gNbIterations );

   // Refracted ray
   float4 reflectedOrigins[gNbIterations];
   float4 reflectedDirections[gNbIterations];
   int    reflectedRays(0);
   float4 reflectedColor = {0.f,0.f,0.f,0.f};
   float  reflectedRatio = 0.f;

   // Variable declarations
   float  shadowIntensity = 0.f;
   float4 refractionFromColor;
   float4 reflectedTarget;

   while( iteration<gNbIterations && carryon ) 
   {
      // Compute intesection with lamps
      if( intersectionWithLamps( lamps, nbActiveLamps, rayOrigin, rayTarget, intersectionColor ) )
      {
         recursiveColor[iteration] = intersectionColor;
         carryon = false;
      }
      else
      {
         carryon = true;
      }

      // If no intersection with lamps detected. Now compute intersection with Primitives
      if( carryon ) 
      {
         carryon = intersectionWithPrimitives(
            primitives, nbActivePrimitives,
            materials, textures, levels,
            rayOrigin, rayTarget,
            timer, 
            closestPrimitive, closestIntersection, 
            normal,
            transparentColor);
      }

      if( carryon ) 
      {
         if ( iteration==0 )
         {
            firstIntersection = closestIntersection;
         }

         // Get object color
         recursiveColor[iteration] = colorFromObject( 
            primitives, nbActivePrimitives, lamps, nbActiveLamps, materials, textures, kinectVideo, levels,
            origin, normal, closestPrimitive, closestIntersection, 
            timer, refractionFromColor, shadowIntensity, recursiveBlinn[iteration], transparentColor );

         // ----------
         // Refraction
         // ----------
         if( materials[primitives[closestPrimitive].materialId].transparency != 0.f ) 
         {
            // ----------
            // Refraction
            // ----------
            // Replace the normal using the intersection color
            // r,g,b become x,y,z... What the fuck!!
            if( materials[primitives[closestPrimitive].materialId].texture.y != NO_TEXTURE) 
            {
               refractionFromColor -= 0.5f;
               normal *= refractionFromColor;
            }

            O_E = rayOrigin - closestIntersection;
            float refraction = materials[primitives[closestPrimitive].materialId].refraction;
            refraction = (refraction == initialRefraction) ? 1.0f : refraction;
            vectorRefraction( O_R, O_E, refraction, normal, initialRefraction );
            reflectedTarget = closestIntersection - O_R;
            initialRefraction = refraction;

            recursiveRatio[iteration].x = materials[primitives[closestPrimitive].materialId].transparency;
            recursiveRatio[iteration].z = 1.f;
         }

         // ----------
         // Reflection
         // ----------
         if( reflectedRays == 0 && materials[primitives[closestPrimitive].materialId].reflection != 0.f ) 
         {
            reflectedRatio = materials[primitives[closestPrimitive].materialId].reflection;
            O_E = rayOrigin - closestIntersection;
            vectorReflection( O_R, O_E, normal );

            reflectedOrigins[reflectedRays]    = closestIntersection; 
            reflectedDirections[reflectedRays] = closestIntersection - O_R;
            reflectedRays++;
         }
         rayOrigin = closestIntersection; 
         rayTarget = reflectedTarget;

         iteration++; 
      }
   }

   for( int i(0); i<reflectedRays; ++i )
   {
      carryon = intersectionWithPrimitives(
         primitives, nbActivePrimitives,
         materials, textures, levels,
         reflectedOrigins[i], reflectedDirections[i],
         timer, 
         closestPrimitive, closestIntersection, 
         normal,
         transparentColor);
      if( carryon )
      {
         // Get object color
         reflectedColor = colorFromObject( 
            primitives, nbActivePrimitives, lamps, nbActiveLamps, materials, textures, kinectVideo, levels,
            origin, normal, closestPrimitive, closestIntersection, 
            timer, refractionFromColor, shadowIntensity, recursiveBlinn[iteration], transparentColor );
      }
   }

   for( int i=iteration-2; i>=0; --i ) 
   {
      recursiveColor[i] = (recursiveColor[i+1]*recursiveRatio[i].x + recursiveColor[i]*(1.f-recursiveRatio[i].x));
   }
   intersectionColor = recursiveColor[0]*(1.f-reflectedRatio) + reflectedColor*reflectedRatio;

   // Specular reflection
   intersectionColor += recursiveBlinn[0];

   saturateVector( intersectionColor );
   intersection = closestIntersection;

   // --------------------------------------------------
   // Attenation effect (Fog)
   // --------------------------------------------------
   float4 O_I = firstIntersection - origin;
   float len = 1.f-(vectorLength(O_I)/gMaxViewDistance);
   len = (len>0.f) ? len : 0.f; 
   intersectionColor.x = intersectionColor.x * len;
   intersectionColor.y = intersectionColor.y * len;
   intersectionColor.z = intersectionColor.z * len;

   // Depth of field
   float4 FI_I = firstIntersection - target;
   float dof = (vectorLength(FI_I)-depthOfField)/gMaxViewDistance;
   depthOfField = dof; 
   return intersectionColor;
}


/**
* ________________________________________________________________________________
* Main Kernel!!!
* ________________________________________________________________________________
*/
__global__ void render( 
   Primitive* primitives, 
   int	     nbActivePrimitives,
   Lamp*      lamps,
   int		  nbActiveLamps,
   Material*  materials,
   char*      textures,
   char*      kinectVideo,
   int*       levels,
   float4     origin,
   float4     target,
   float4     angles,
   int        width, 
   int        height, 
   float      pointOfFocus,
   int        draft,
   float      transparentColor,
   float      timer,
   float4*    depthOfField,
   char*      bitmap)
{
   int x = blockDim.x*blockIdx.x + threadIdx.x;
   int y = blockDim.y*blockIdx.y + threadIdx.y;
   int index = y*width+x;

   float4 rotationCenter = {0.f,0.f,0.f,0.f};
   vectorRotation( origin, rotationCenter, angles );

   depthOfField[index].x = 0.f;
   depthOfField[index].y = 0.f;
   depthOfField[index].z = 0.f;
   depthOfField[index].w = 0.f;

   float dof = pointOfFocus;

   target.x = target.x - 2.f*(float)(x - (width/2));
   target.y = target.y + 2.f*(float)(y - (height/2));
   vectorRotation( target, rotationCenter, angles );

   float4 intersection;
   depthOfField[index] = launchRay(
      primitives, nbActivePrimitives,
      lamps, nbActiveLamps,
      materials, textures, kinectVideo, levels,
      origin, target, timer, 
      transparentColor,
      intersection,
      dof);
   depthOfField[index].w = dof;

   //makeOpenGLColor( depthOfField[index], bitmap, index ); 
}


__global__ void postProcess(
   int width, int height,
   float4* depthOfField,
   float*  randoms,
   char*   bitmap) 
{
   int x = blockDim.x*blockIdx.x + threadIdx.x;
   int y = blockDim.y*blockIdx.y + threadIdx.y;
   int index = y*width+x;
#if 1
   float  depth = 30.f*depthOfField[index].w;
   int    wh = width*height;

   float4 localColor;
   localColor.x = 0.f;
   localColor.y = 0.f;
   localColor.z = 0.f;

   for( int i=0; i<100; ++i )
   {
      int ix = (int(i+depthOfField))%wh;
      int iy = (i+width)%wh;
      int xx = x+depth*randoms[ix];
      int yy = y+depth*randoms[iy];
      if( xx>=0 && xx<width && yy>=0 && yy<height )
      {
         int localIndex = yy*width+xx;
         if( localIndex>=0 && localIndex<wh )
         {
            localColor += depthOfField[localIndex];
         }
      }
   }
   localColor /= 100.f;
   localColor.w = 0.f;

   makeOpenGLColor( localColor, bitmap, index ); 
#else
   makeOpenGLColor( depthOfField[index], bitmap, index ); 
#endif
}

extern "C" void initialize_scene( 
   int width, int height, int nbPrimitives, int nbLamps, int nbMaterials, int nbTextures, int nbLevels )
{
   cutilSafeCall(hipMalloc( (void**)&d_primitives,  nbPrimitives*sizeof(Primitive)));
   cutilSafeCall(hipMalloc( (void**)&d_lamps,       nbLamps*sizeof(Lamp)));
   cutilSafeCall(hipMalloc( (void**)&d_materials,   nbMaterials*sizeof(Material)));
   cutilSafeCall(hipMalloc( (void**)&d_textures,    nbTextures*gTextureDepth*gTextureWidth*gTextureHeight));
   cutilSafeCall(hipMalloc( (void**)&d_randoms,     width*height*sizeof(float)));
   cutilSafeCall(hipMalloc( (void**)&d_levels,      nbLevels*sizeof(int)));
   cutilSafeCall(hipMalloc( (void**)&d_depthOfField,width*height*sizeof(float4)));
   cutilSafeCall(hipMalloc( (void**)&d_bitmap,      width*height*gColorDepth*sizeof(char)));
   cutilSafeCall(hipMalloc( (void**)&d_kinectVideo, gKinectVideo*gKinectVideoWidth*gKinectVideoHeight*sizeof(char)));
   cutilSafeCall(hipMalloc( (void**)&d_kinectDepth, gKinectDepth*gKinectDepthWidth*gKinectDepthHeight*sizeof(char)));
}

extern "C" void finalize_scene()
{
   cutilSafeCall(hipFree( d_primitives ));
   cutilSafeCall(hipFree( d_lamps ));
   cutilSafeCall(hipFree( d_materials ));
   cutilSafeCall(hipFree( d_textures ));
   cutilSafeCall(hipFree( d_randoms ));
   cutilSafeCall(hipFree( d_levels ));
   cutilSafeCall(hipFree( d_depthOfField ));
   cutilSafeCall(hipFree( d_bitmap ));
   cutilSafeCall(hipFree( d_kinectVideo ));
   cutilSafeCall(hipFree( d_kinectDepth ));
}

extern "C" void h2d_scene( 
   Primitive*  primitives, int nbPrimitives,
   Lamp*       lamps,      int nbLamps )
{
   cutilSafeCall(hipMemcpy( d_primitives, primitives, nbPrimitives*sizeof(Primitive), hipMemcpyHostToDevice ));
   cutilSafeCall(hipMemcpy( d_lamps,      lamps,      nbLamps*sizeof(Lamp),           hipMemcpyHostToDevice ));
}

extern "C" void h2d_materials( 
   Material*  materials, int nbActiveMaterials,
   char*      textures , int nbActiveTextures,
   float*     randoms,   int nbRandoms,
   int*       levels,    int levelSize)
{
   cutilSafeCall(hipMemcpy( d_materials, materials, nbActiveMaterials*sizeof(Material), hipMemcpyHostToDevice ));
   cutilSafeCall(hipMemcpy( d_textures,  textures,  nbActiveTextures*sizeof(char)*gTextureDepth*gTextureWidth*gTextureHeight,  hipMemcpyHostToDevice ));
   cutilSafeCall(hipMemcpy( d_randoms,   randoms,   nbRandoms*sizeof(float), hipMemcpyHostToDevice ));
   cutilSafeCall(hipMemcpy( d_levels,    levels,    levelSize*sizeof(int), hipMemcpyHostToDevice ));
}

extern "C" void d2h_bitmap( unsigned char* bitmap, int size )
{
   cutilSafeCall(hipMemcpy( bitmap, d_bitmap, size, hipMemcpyDeviceToHost ));
}

extern "C" void h2d_kinect( 
   char* kinectVideo, int videoSize,
   char* kinectDepth, int depthSize )
{
   cutilSafeCall(hipMemcpy( d_kinectVideo, kinectVideo, videoSize*sizeof(char), hipMemcpyHostToDevice ));
   cutilSafeCall(hipMemcpy( d_kinectDepth, kinectDepth, depthSize*sizeof(char), hipMemcpyHostToDevice ));
}

/**
* @brief Run the kernel on the GPU. 
* This function is executed on the Host.
*/
extern "C" void cudaRender(
   dim3 blockSize,
   int nbPrimitives, int nbLamps,
   float4 origin, float4 target, float4 angles,
   int width, int height, 
   float pointOfFocus, int draft,
   float transparentColor, float timer)
{
   // Run the Kernel
   dim3 grid((width+blockSize.x-1)/blockSize.x,(height+blockSize.y-1)/blockSize.y,1);

   render<<<grid,blockSize>>>(
      d_primitives, nbPrimitives, 
      d_lamps, nbLamps,
      d_materials, d_textures, d_kinectVideo, d_levels,
      origin, target, angles, width, height, 
      pointOfFocus, draft, 
      transparentColor, timer,
      d_depthOfField, d_bitmap );

   postProcess<<<grid,blockSize>>>(
      width, height, d_depthOfField, d_randoms, d_bitmap );
}
